#include <hipper/hipper_runtime.h>
#include <hipper/hipper_cub.h>
#include "test_hipper.h"

TEST_CASE("CUB operations", "[CUB]")
    {
    // input (array of integers)
    int* a;
    const int N = 2;
    REQUIRE_SUCCESS(hipper::mallocManaged(reinterpret_cast<void**>(&a), sizeof(int)*N));
    a[0] = 1;
    a[1] = 2;

    // output (sum of a)
    int* total;
    REQUIRE_SUCCESS(hipper::mallocManaged(reinterpret_cast<void**>(&total), sizeof(int)));
    *total = 0;

    // size temporary memory
    void *tmp = NULL;
    size_t tmp_bytes = 0;
    hipper::hipcub::DeviceReduce::Sum(tmp,tmp_bytes,a,total,N);
    REQUIRE_SUCCESS(hipper::mallocManaged(reinterpret_cast<void**>(&tmp), tmp_bytes));

    // take sum
    hipper::hipcub::DeviceReduce::Sum(tmp,tmp_bytes,a,total,N);
    REQUIRE_SUCCESS(hipper::deviceSynchronize());

    // check output
    REQUIRE(a[0] == 1);
    REQUIRE(a[1] == 2);
    REQUIRE(*total == 3);

    // free memory
    REQUIRE_SUCCESS(hipper::free(a));
    REQUIRE_SUCCESS(hipper::free(total));
    REQUIRE_SUCCESS(hipper::free(tmp));
    }
