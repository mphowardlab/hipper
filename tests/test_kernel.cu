#include "hip/hip_runtime.h"
#include <hipper/hipper_runtime.h>
#include "test_hipper.h"

__global__ void set_thread(int* a, const int N)
    {
    int rank = hipper::threadRank<1,1>();
    if (rank >= N) return;

    a[rank] = rank+1;
    }

TEST_CASE("Basic kernel launch", "[kernel]")
    {
    int* a;
    const int N = 2;

    REQUIRE_SUCCESS(hipper::mallocManaged(reinterpret_cast<void**>(&a), sizeof(int)*N));

    REQUIRE_SUCCESS(hipper::memset(a, 0, sizeof(int)*N));
    REQUIRE(a[0] == 0);
    REQUIRE(a[1] == 0);

    hipper::KernelLauncher(1, 32)(set_thread, a, N);
    REQUIRE_SUCCESS(hipper::peekAtLastError());
    REQUIRE_SUCCESS(hipper::deviceSynchronize());
    REQUIRE(a[0] == 1);
    REQUIRE(a[1] == 2);

    REQUIRE_SUCCESS(hipper::free(a));
    }
